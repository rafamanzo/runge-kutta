#include<stdlib.h>
#include<math.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../vector_field.h"

__device__ int offset(int n_x, int n_y, int x, int y, int z){
  return x + n_x*y + n_y*n_x*z;
}
