#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<math.h>
#include "../vector_field.h"

__device__ vector sum(vector v1, vector v2){
  vector sum;
  
  sum.x = v1.x + v2.x;
  sum.y = v1.y + v2.y;
  sum.z = v1.z + v2.z;
  
  return sum;
}

__device__ vector mult_scalar(vector v, double scalar){
  vector mult;
  
  mult.x = v.x*scalar;
  mult.y = v.y*scalar;
  mult.z = v.z*scalar;
  
  return mult;
}

__device__ void set(vector *x, vector y){
  (*x).x = y.x;
  (*x).y = y.y;
  (*x).z = y.z;
}

__device__ double module(vector v){
  return sqrt(pow(v.x, 2) + pow(v.y, 2) + pow(v.z, 2));
}

__device__ double distance(vector x, vector y){
  return module(sum(x, mult_scalar(y, -1.0)));
}
