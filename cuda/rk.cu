#include "hip/hip_runtime.h"
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../first_order_function.h"
#include "rk_kernel.h"

__global__ void rk2_kernel(double *x0, double *y0, double h, fof *dydx, double *y1){
  __shared__ double k1, k2;
  __shared__ int i = threadIdx.x;
  
  k1 = h*(dydx[i].a*x0[i] + dydx[i].b*y0[i] + dydx[i].c);
  k2 = h*(dydx[i].a*(x0[i] + h/2.0) + dydx[i].b*(y0[i] + k1/2.0) + dydx[i].c);
  y1[i] = y0[i] + k2;
}


__global__ void rk4_kernel(double *x0, double *y0, double h, fof *dydx, double *y1){
  __shared__ double k1, k2, k3, k4;
  __shared__ int i = threadIdx.x;
  
  k1 = h*(dydx[i].a*x0[i] + dydx[i].b*y0[i] + dydx[i].c);
  k2 = h*(dydx[i].a*(x0[i] + h/2.0) + dydx[i].b*(y0[i] + k1/2.0) + dydx[i].c);
  k3 = h*(dydx[i].a*(x0[i] + h/2.0) + dydx[i].b*(y0[i] + k2/2.0) + dydx[i].c);
  k4 = h*(dydx[i].a*(x0[i] + h) + dydx[i].b*(y0[i] + k3) + dydx[i].c);
  y1[i] = y0[i] + k1/6.0 + k2/3.0 + k3/3.0 + k4/6.0;
}

void rk2_caller(double *x0, double *y0, int n, double h, fof *dydx, double *y1){
  double *d_x0;
  double *d_y0;
  fof *d_dydx;
  double *d_y1;
  
  hipMalloc(&d_x0, n*sizeof(double));
  hipMalloc(&d_y0, n*sizeof(double));
  hipMalloc(&d_dydx, n*sizeof(fof));
  hipMalloc(&d_y1, n*sizeof(double));
  
  hipMemcpy(d_x0, x0, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_y0, y0, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dydx, dydx, n*sizeof(fof), hipMemcpyHostToDevice);
  
  rk2_kernel<<<1,n>>>(d_x0, d_y0, h, d_dydx, d_y1);
  
  hipMemcpy(y1, d_y1, n*sizeof(double), hipMemcpyDeviceToHost);
}

void rk4_caller(double *x0, double *y0, int n, double h, fof *dydx, double *y1){
  double *d_x0;
  double *d_y0;
  fof *d_dydx;
  double *d_y1;
  
  hipMalloc(&d_x0, n*sizeof(double));
  hipMalloc(&d_y0, n*sizeof(double));
  hipMalloc(&d_dydx, n*sizeof(fof));
  hipMalloc(&d_y1, n*sizeof(double));
  
  hipMemcpy(d_x0, x0, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_y0, y0, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dydx, dydx, n*sizeof(fof), hipMemcpyHostToDevice);
  
  rk4_kernel<<<1,n>>>(d_x0, d_y0, h, d_dydx, d_y1);
  
  hipMemcpy(y1, d_y1, n*sizeof(double), hipMemcpyDeviceToHost);
}
