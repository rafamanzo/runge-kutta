#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<math.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../vector_field.h"
#include "vector_operations.h"
#include "rk_kernel.h"

__device__ vector nearest_neighbour(vector v0, int n_x, int n_y, int n_z, vector_field field){
  int x, y, z;
  vector zero;
  
  zero.x = zero.y = zero.z = 0.0;
  
  if( (v0.x - floor(v0.x)) > 0.5 && v0.x < (n_x - 1))
    x = (int) ceil(v0.x);
  else
    x = (int) floor(v0.x);
    
  if( (v0.y - floor(v0.y)) > 0.5 && v0.y < (n_y - 1))
    y = (int) ceil(v0.y);
  else
    y = (int) floor(v0.y);
    
  if( (v0.z - floor(v0.z)) > 0.5 && v0.z < (n_z - 1))
    z = (int) ceil(v0.z);
  else
    z = (int) floor(v0.z);

  if(x >= n_x || y >= n_y || z >= n_z || x < 0 || y < 0 || z < 0){
    return zero;
  }else{
    return field[offset(n_x, n_y, x, y, z)];
  }
}

__device__ vector trilinear_interpolation(vector v0, int n_x, int n_y, int n_z, vector_field field){
  int x1, y1, z1, x0, y0, z0, xd, yd, zd;
  vector zero, i1, i2, j1, j2, w1, w2;
  
  zero.x = zero.y = zero.z = 0.0;
  
  x1 = ceil(v0.x);
  y1 = ceil(v0.y);
  z1 = ceil(v0.z);
  x0 = floor(v0.x);
  y0 = floor(v0.y);
  z0 = floor(v0.z);
  
  if(x1 >= n_x || y1 >= n_y || z1 >= n_z || x0 < 0 || y0 < 0 || z0 < 0){
    return nearest_neighbour(v0, n_x, n_y, n_z, field);
  }else{
    xd = (v0.x - x0)/(x1 - x0);
    yd = (v0.y - y0)/(y1 - y0);
    zd = (v0.z - z0)/(z1 - z0);
    
    set(&i1, sum( mult_scalar(field[offset(n_x, n_y, x0, y0, z0)], (1.0 - zd)), mult_scalar(field[offset(n_x, n_y, x0, y0, z1)], zd) ) );
    set(&i2, sum( mult_scalar(field[offset(n_x, n_y, x0, y1, z0)], (1.0 - zd)), mult_scalar(field[offset(n_x, n_y, x0, y1, z1)], zd) ) );
    set(&j1, sum( mult_scalar(field[offset(n_x, n_y, x1, y0, z0)], (1.0 - zd)), mult_scalar(field[offset(n_x, n_y, x1, y0, z1)], zd) ) );
    set(&j2, sum( mult_scalar(field[offset(n_x, n_y, x1, y1, z0)], (1.0 - zd)), mult_scalar(field[offset(n_x, n_y, x1, y1, z1)], zd) ) );
    
    set(&w1, sum( mult_scalar(i1, (1.0 - yd)), mult_scalar(i2, yd) ) );
    set(&w2, sum( mult_scalar(j1, (1.0 - yd)), mult_scalar(j2, yd) ) );
    
    return sum( mult_scalar(w1, (1.0 - xd)), mult_scalar(w2, xd) );
  }
}

__global__ void rk2_kernel(double *x0, double *y0, double h, fof *dydx, double *y1){
  __shared__ double k1, k2;
  __shared__ int i;

  i = threadIdx.x;
  
  k1 = h*(dydx[i].a*x0[i] + dydx[i].b*y0[i] + dydx[i].c);
  k2 = h*(dydx[i].a*(x0[i] + h/2.0) + dydx[i].b*(y0[i] + k1/2.0) + dydx[i].c);
  y1[i] = y0[i] + k2;
}


/*__global__ void rk4_kernel(double *x0, double *y0, double h, fof *dydx, double *y1){
  __shared__ double k1, k2, k3, k4;
  __shared__ int i;
  
  i = threadIdx.x;
  
  k1 = h*(dydx[i].a*x0[i] + dydx[i].b*y0[i] + dydx[i].c);
  k2 = h*(dydx[i].a*(x0[i] + h/2.0) + dydx[i].b*(y0[i] + k1/2.0) + dydx[i].c);
  k3 = h*(dydx[i].a*(x0[i] + h/2.0) + dydx[i].b*(y0[i] + k2/2.0) + dydx[i].c);
  k4 = h*(dydx[i].a*(x0[i] + h) + dydx[i].b*(y0[i] + k3) + dydx[i].c);
  y1[i] = y0[i] + k1/6.0 + k2/3.0 + k3/3.0 + k4/6.0;
}*/

void rk2_caller(vector *v0, int count_v0, double h, int n_x, int n_y, int n_z, vector_field field, vector ***points, int **n_points){
  vector *d_v0;
  vector_field d_field;
  vector **d_points;
  int *d_n_points;
  
  hipMalloc(&d_v0, count_v0*sizeof(vector));
  hipMalloc(&d_field, n_x*n_y*n_z*sizeof(vector));
  hipMalloc(&d_points, count_v0*sizeof(vector *));
  hipMalloc(&d_n_points, count_v0*sizeof(int));
  
  hipMemcpy(d_v0, v0, count_v0*sizeof(vector), hipMemcpyHostToDevice);
  hipMemcpy(d_field, field, n_x*n_y*n_z*sizeof(vector), hipMemcpyHostToDevice);
  
  rk2_kernel<<<1,n>>>(d_v0, count_v0, h, n_x, n_y, n_z, d_field, d_points, d_n_points);
  
  hipMemcpy(y1, d_y1, n*sizeof(double), hipMemcpyDeviceToHost);
  
  hipFree(d_x0);
  hipFree(d_y0);
  hipFree(d_y1);
  hipFree(d_dydx);
}

/*void rk4_caller(double *x0, double *y0, int n, double h, fof *dydx, double *y1){
  double *d_x0;
  double *d_y0;
  fof *d_dydx;
  double *d_y1;
  
  hipMalloc(&d_x0, n*sizeof(double));
  hipMalloc(&d_y0, n*sizeof(double));
  hipMalloc(&d_dydx, n*sizeof(fof));
  hipMalloc(&d_y1, n*sizeof(double));
  
  hipMemcpy(d_x0, x0, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_y0, y0, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_dydx, dydx, n*sizeof(fof), hipMemcpyHostToDevice);
  
  rk4_kernel<<<1,n>>>(d_x0, d_y0, h, d_dydx, d_y1);
  
  hipMemcpy(y1, d_y1, n*sizeof(double), hipMemcpyDeviceToHost);
  
  hipFree(d_x0);
  hipFree(d_y0);
  hipFree(d_y1);
  hipFree(d_dydx);
}*/
