#include "hip/hip_runtime.h"
#include<cstdlib>
#include<cstdio>
#include<cmath>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include<dataset.h>
#include<fiber.h>
#include<rk_cuda_kernel.h>

/******************************/
/* Auxiliary Vector Functions */
/******************************/

/*FIXME: there must be libraries inside CUDA to work with vectors*/

__device__ int cuda_offset(int n_x, int n_y, int x, int y, int z){
  return x + n_x*y + n_y*n_x*z;
}

__device__ vector sum(vector v1, vector v2){
  vector sum;
  
  sum.x = v1.x + v2.x;
  sum.y = v1.y + v2.y;
  sum.z = v1.z + v2.z;
  
  return sum;
}

__device__ vector mult_scalar(vector v, double scalar){
  vector mult;
  
  mult.x = v.x*scalar;
  mult.y = v.y*scalar;
  mult.z = v.z*scalar;
  
  return mult;
}

__device__ void set(vector *x, vector y){
  (*x).x = y.x;
  (*x).y = y.y;
  (*x).z = y.z;
}

__device__ double module(vector v){
  return sqrt(pow(v.x, 2) + pow(v.y, 2) + pow(v.z, 2));
}

__device__ double distance(vector x, vector y){
  return module(sum(x, mult_scalar(y, -1.0)));
}

/************************************/
/* Auxiliary Aproximation Functions */
/************************************/

__device__ vector nearest_neighbour(vector v0, int n_x, int n_y, int n_z, vector_field field){
  int x, y, z;
  vector zero;
  
  zero.x = zero.y = zero.z = 0.0;
  
  if( (v0.x - floor(v0.x)) > 0.5 && v0.x < (n_x - 1))
    x = (int) ceil(v0.x);
  else
    x = (int) floor(v0.x);
    
  if( (v0.y - floor(v0.y)) > 0.5 && v0.y < (n_y - 1))
    y = (int) ceil(v0.y);
  else
    y = (int) floor(v0.y);
    
  if( (v0.z - floor(v0.z)) > 0.5 && v0.z < (n_z - 1))
    z = (int) ceil(v0.z);
  else
    z = (int) floor(v0.z);

  if(x >= n_x || y >= n_y || z >= n_z || x < 0 || y < 0 || z < 0){
    return zero;
  }else{
    return field[cuda_offset(n_x, n_y, x, y, z)];
  }
}

__device__ vector trilinear_interpolation(vector v0, int n_x, int n_y, int n_z, vector_field field){
  int x1, y1, z1, x0, y0, z0, xd, yd, zd;
  vector i1, i2, j1, j2, w1, w2;
  
  x1 = ceil(v0.x);
  y1 = ceil(v0.y);
  z1 = ceil(v0.z);
  x0 = floor(v0.x);
  y0 = floor(v0.y);
  z0 = floor(v0.z);
  
  if(x1 >= n_x || y1 >= n_y || z1 >= n_z || x0 < 0 || y0 < 0 || z0 < 0){
    return nearest_neighbour(v0, n_x, n_y, n_z, field);
  }else{
    xd = (v0.x - x0)/(x1 - x0);
    yd = (v0.y - y0)/(y1 - y0);
    zd = (v0.z - z0)/(z1 - z0);
    
    set(&i1, sum( mult_scalar(field[cuda_offset(n_x, n_y, x0, y0, z0)], (1.0 - zd)), mult_scalar(field[cuda_offset(n_x, n_y, x0, y0, z1)], zd) ) );
    set(&i2, sum( mult_scalar(field[cuda_offset(n_x, n_y, x0, y1, z0)], (1.0 - zd)), mult_scalar(field[cuda_offset(n_x, n_y, x0, y1, z1)], zd) ) );
    set(&j1, sum( mult_scalar(field[cuda_offset(n_x, n_y, x1, y0, z0)], (1.0 - zd)), mult_scalar(field[cuda_offset(n_x, n_y, x1, y0, z1)], zd) ) );
    set(&j2, sum( mult_scalar(field[cuda_offset(n_x, n_y, x1, y1, z0)], (1.0 - zd)), mult_scalar(field[cuda_offset(n_x, n_y, x1, y1, z1)], zd) ) );
    
    set(&w1, sum( mult_scalar(i1, (1.0 - yd)), mult_scalar(i2, yd) ) );
    set(&w2, sum( mult_scalar(j1, (1.0 - yd)), mult_scalar(j2, yd) ) );
    
    return sum( mult_scalar(w1, (1.0 - xd)), mult_scalar(w2, xd) );
  }
}

/***********/
/* Kernels */
/***********/

__global__ void rk2_kernel(vector *v0, int count_v0, double h, int n_x, int n_y, int n_z, vector_field field, vector *points, int *n_points, int max_points){
  /*TODO: moving the field to the shared memory should increase performance*/
  vector k1, k2, initial, direction;
  int i, n_points_aux;
  
  n_points_aux = 0;
  
  i = threadIdx.x;
  
  set( &initial, v0[i] );
  set( &direction, field[cuda_offset(n_x, n_y, initial.x, initial.y, initial.z)] );
  
  while(floor(module(direction)) > 0.0 && n_points_aux < max_points){
    n_points_aux++;
        
    set( &(points[cuda_offset(count_v0, 0, i, n_points_aux - 1, 0)]), initial );
  
    set( &k1, mult_scalar( direction, h ) );
    set( &k2, sum( mult_scalar(k1, 0.5), mult_scalar( direction, h ) ) );
    
    set( &initial, sum( initial, k2) );
    set( &direction, trilinear_interpolation(initial, n_x, n_y, n_z, field) );
  }
  
  n_points[i] = n_points_aux;
  n_points_aux = 0;
}

__global__ void rk4_kernel(vector *v0, int count_v0, double h, int n_x, int n_y, int n_z, vector_field field, vector *points, int *n_points, int max_points){
  /*TODO: moving the field to the shared memory should increase performance*/
  vector k1, k2, k3, k4, initial, direction;
  int i, n_points_aux;
  
  n_points_aux = 0;
  
  i = threadIdx.x;
  
  set( &initial, v0[i] );
  set( &direction, field[cuda_offset(n_x, n_y, initial.x, initial.y, initial.z)] );
  
  while(floor(module(direction)) > 0.0 && n_points_aux < max_points){
    n_points_aux++;
        
    set( &(points[cuda_offset(count_v0, 0, i, n_points_aux - 1, 0)]), initial );
  
    set( &k1, mult_scalar( direction, h ) );
    set( &k2, sum( mult_scalar(k1, 0.5), mult_scalar( direction, h ) ) );
    set( &k3, sum( mult_scalar(k2, 0.5), mult_scalar( direction, h ) ) );
    set( &k4, sum( k3, mult_scalar( direction, h ) ) );
    
    set( &initial, sum( initial, sum( mult_scalar( k1 , 1.0/6.0 ), sum( mult_scalar( k2, 1.0/3.0 ), sum( mult_scalar( k3, 1.0/3.0 ), mult_scalar( k4, 1.0/6.0 ) ) ) ) ) );
    set( &direction, trilinear_interpolation(initial, n_x, n_y, n_z, field) );
  }
  
  n_points[i] = n_points_aux;
}

/***********/
/* Callers */
/***********/

void rk2_caller(vector *v0, int count_v0, double h, int n_x, int n_y, int n_z, vector_field field, runge_kutta::Fiber **fibers){
  vector *d_v0;
  vector_field d_field;
  vector *d_points, *points_aux;
  int *d_n_points, *n_points_aux;
  int i, j, max_points;
  hipEvent_t start, finish;
  float time;
  size_t available, total;
    
  hipEventCreate(&start);
  hipEventCreate(&finish);
  
  hipEventRecord(start, 0);
  
  if(hipMalloc(&d_v0, count_v0*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the initial points\n", (count_v0*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  if(hipMalloc(&d_field, n_x*n_y*n_z*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the vector field\n", (n_x*n_y*n_z*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  if(hipMalloc(&d_n_points, count_v0*sizeof(int)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the points count vector\n", (count_v0*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  hipMemGetInfo(&available, &total);
  max_points = ((available*0.9)/(sizeof(vector)*count_v0));
  if(hipMalloc(&d_points, count_v0*max_points*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the fibers\n", (count_v0*max_points*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }  
  hipMemcpy(d_v0, v0, count_v0*sizeof(vector), hipMemcpyHostToDevice);
  hipMemcpy(d_field, field, n_x*n_y*n_z*sizeof(vector), hipMemcpyHostToDevice);
  
  /*TODO: adjust threads per block to maximize performance*/
  rk2_kernel<<<1,count_v0>>>(d_v0, count_v0, h, n_x, n_y, n_z, d_field, d_points, d_n_points, max_points);
  hipDeviceSynchronize();
  
  n_points_aux = (int *) malloc(count_v0*sizeof(int));
  points_aux = (vector *) malloc(count_v0*max_points*sizeof(vector));;
  
  hipMemcpy(n_points_aux, d_n_points, count_v0*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(points_aux, d_points, count_v0*max_points*sizeof(vector), hipMemcpyDeviceToHost);
  
  hipEventRecord(finish, 0);
  hipEventSynchronize(finish);
  
  hipEventElapsedTime(&time, start, finish);
  hipEventDestroy(start);
  hipEventDestroy(finish);
  
  printf("GPU time for RK2: %fs\n", time/1000.0);
  
  *fibers = (runge_kutta::Fiber *) malloc(count_v0*sizeof(runge_kutta::Fiber));
  for(i = 0; i < count_v0; i++){
    (*fibers)[i] = runge_kutta::Fiber(n_points_aux[i]);
    for(j = 0; j < n_points_aux[i]; j++){
      (*fibers)[i].setPoint(j, points_aux[runge_kutta::DataSet::offset(count_v0, 0, i, j, 0)]);
    }
  }
  
  hipFree(d_v0);
  hipFree(d_field);
  hipFree(d_points);
  hipFree(d_n_points);
}

void rk4_caller(vector *v0, int count_v0, double h, int n_x, int n_y, int n_z, vector_field field, runge_kutta::Fiber **fibers){
  vector *d_v0;
  vector_field d_field;
  vector *d_points, *points_aux;
  int *d_n_points, *n_points_aux;
  int i, j, max_points;
  hipEvent_t start, finish;
  float time;
  size_t available, total;

  hipEventCreate(&start);
  hipEventCreate(&finish);
  
  hipEventRecord(start, 0);
  
  if(hipMalloc(&d_v0, count_v0*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the initial points\n", (count_v0*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  if(hipMalloc(&d_field, n_x*n_y*n_z*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the vector field\n", (n_x*n_y*n_z*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  if(hipMalloc(&d_n_points, count_v0*sizeof(int)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the points count vector\n", (count_v0*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  hipMemGetInfo(&available, &total);
  max_points = ((available*0.9)/(sizeof(vector)*count_v0));
  if(hipMalloc(&d_points, count_v0*max_points*sizeof(vector)) == hipErrorOutOfMemory){
    printf("\nCould not allocate %fMB for the fibers\n", (count_v0*max_points*sizeof(vector))/1024.0/1024.0);
    exit(-1);
  }
  
  hipMemcpy(d_v0, v0, count_v0*sizeof(vector), hipMemcpyHostToDevice);
  hipMemcpy(d_field, field, n_x*n_y*n_z*sizeof(vector), hipMemcpyHostToDevice);
  
  
  /*TODO: adjust threads per block to maximize performance*/
  rk4_kernel<<<1,count_v0>>>(d_v0, count_v0, h, n_x, n_y, n_z, d_field, d_points, d_n_points, max_points);
  hipDeviceSynchronize();
  
  n_points_aux = (int *) malloc(count_v0*sizeof(int));
  points_aux = (vector *) malloc(count_v0*max_points*sizeof(vector));
  
  hipMemcpy(n_points_aux, d_n_points, count_v0*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(points_aux, d_points, count_v0*max_points*sizeof(vector), hipMemcpyDeviceToHost);
  
  hipEventRecord(finish, 0);
  hipEventSynchronize(finish);
  
  hipEventElapsedTime(&time, start, finish);
  hipEventDestroy(start);
  hipEventDestroy(finish);
  
  printf("GPU time for RK4: %fs\n", time/1000.0);
  
  *fibers = (runge_kutta::Fiber *) malloc(count_v0*sizeof(runge_kutta::Fiber));
  for(i = 0; i < count_v0; i++){
    (*fibers)[i] = runge_kutta::Fiber(n_points_aux[i]);
    for(j = 0; j < n_points_aux[i]; j++){
      (*fibers)[i].setPoint(j, points_aux[runge_kutta::DataSet::offset(count_v0, 0, i, j, 0)]);
    }
  }
  
  hipFree(d_v0);
  hipFree(d_field);
  hipFree(d_points);
  hipFree(d_n_points);
}
